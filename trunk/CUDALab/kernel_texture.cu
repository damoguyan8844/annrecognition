#include "hip/hip_runtime.h"
#define BLOCK_DIM 512
texture<unsigned char, 1, hipReadModeElementType> rT1;
texture<unsigned char, 1, hipReadModeElementType> rT2;

extern "C" void Blend_GPU_Texture( unsigned char* aImg1, unsigned char* aImg2, unsigned char* aRS, int width, int height );
extern "C" void Blend_GPU_Texture_kernel_only( unsigned char* aRS, int size );
extern "C" void BindTexture( unsigned char* aImg1, unsigned char* aImg2 );
extern "C" void UnbindTexture();

__global__ void Blending_Texture_Kernel( unsigned char* aRS, int size )
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if( index < size )
        aRS[index]  = 0.5 * tex1Dfetch( rT1, index ) + 0.5 * tex1Dfetch( rT2, index );
}

void Blend_GPU_Texture( unsigned char* aImg1, unsigned char* aImg2, unsigned char* aRS, int width, int height )
{
    int size = height * width;
    int data_size = size * sizeof( unsigned char );

    // part1, allocate data on device
    unsigned char	*dev_A,	*dev_B,	*dev_C;
    hipMalloc( (void**)&dev_A, data_size );
    hipMalloc( (void**)&dev_B, data_size );
    hipMalloc( (void**)&dev_C, data_size );

    // part2, copy memory to device
    hipMemcpy( dev_A, aImg1, data_size, hipMemcpyHostToDevice );
    hipMemcpy( dev_B, aImg2, data_size, hipMemcpyHostToDevice );

    // part2a, bind texture
    hipBindTexture(0, rT1, dev_A );
    hipBindTexture(0, rT2, dev_B );

    // part3, run kernel
    Blending_Texture_Kernel<<< ceil( (float)size / BLOCK_DIM ), BLOCK_DIM >>>( dev_C, size );

    // part4, copy data from device
    hipMemcpy( aRS, dev_C, data_size, hipMemcpyDeviceToHost );

    // part5, release data
    hipUnbindTexture(rT1);
    hipUnbindTexture(rT2);

    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);
}

void BindTexture( unsigned char* aImg1, unsigned char* aImg2 )
{
    hipBindTexture(0, rT1, aImg1 );
    hipBindTexture(0, rT2, aImg2 );
}

void UnbindTexture()
{
    hipUnbindTexture(rT1);
    hipUnbindTexture(rT2);
}

void Blend_GPU_Texture_kernel_only( unsigned char* aRS, int size )
{
	Blending_Texture_Kernel<<< ceil( (float)size / BLOCK_DIM ), BLOCK_DIM >>>( aRS, size );
}
