
#include <hip/hip_runtime.h>
#define BLOCK_DIM 512

extern "C" void Blend_GPU( unsigned char* aImg1, unsigned char* aImg2, unsigned char* aImg3, int width, int height );
extern "C" void Blend_GPU_kernel_only( unsigned char* aImg1, unsigned char* aImg2, unsigned char* aRS, int size );

__global__ void Blending_Kernel( unsigned char* aR1, unsigned char* aR2, unsigned char* aRS, int size )
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if( index < size )
        aRS[index]  = 0.5 * aR1[index] + 0.5 * aR2[index];
}

void Blend_GPU( unsigned char* aImg1, unsigned char* aImg2, unsigned char* aRS, int width, int height )
{
    int size = height * width;
    int data_size = size * sizeof( unsigned char );

    // part1, allocate data on device
    unsigned char	*dev_A,	*dev_B,	*dev_C;
    hipMalloc( (void**)&dev_A, data_size );
    hipMalloc( (void**)&dev_B, data_size );
    hipMalloc( (void**)&dev_C, data_size );

    // part2, copy memory to device
    hipMemcpy( dev_A, aImg1, data_size, hipMemcpyHostToDevice );
    hipMemcpy( dev_B, aImg2, data_size, hipMemcpyHostToDevice );

    // part3, run kernel
    Blending_Kernel<<< ceil( (float)size / BLOCK_DIM ), BLOCK_DIM >>>( dev_A, dev_B, dev_C, size );

    // part4, copy data from device
    hipMemcpy( aRS, dev_C, data_size, hipMemcpyDeviceToHost );

    // part5, release data
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);
}

void Blend_GPU_kernel_only( unsigned char* aImg1, unsigned char* aImg2, unsigned char* aRS, int size )
{
	Blending_Kernel<<< ceil( (float)size / BLOCK_DIM ), BLOCK_DIM >>>( aImg1, aImg2, aRS, size );
}
