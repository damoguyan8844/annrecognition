#include "hip/hip_runtime.h"
#define BLOCK_DIM 16

texture<unsigned char, 2, hipReadModeElementType> rT;

extern "C" void Transpose_GPU( unsigned char* sImg, unsigned char *tImg, int w, int h );


__global__ void Transpose_Texture( unsigned char* aRS, int w, int h )
{
	int	idxX = blockIdx.x * blockDim.x + threadIdx.x,
		idxY = blockIdx.y * blockDim.y + threadIdx.y;
	if( idxX < w && idxY < h )
		aRS[ idxX * h + idxY ] = tex2D( rT, idxX, idxY );
}

void Transpose_GPU( unsigned char* sImg, unsigned char *tImg, int w, int h )
{
	// compute the size of data
	int	data_size = sizeof(unsigned char) * w * h;

	// part1a. prepare the result data
	unsigned char *dImg;
	hipMalloc( (void**)&dImg, data_size );

	// part1b. prepare the source data
	hipChannelFormatDesc chDesc = hipCreateChannelDesc<unsigned char>();
	hipArray* cuArray;
	hipMallocArray(&cuArray, &chDesc, w, h);
	hipMemcpyToArray( cuArray, 0, 0, sImg, data_size, hipMemcpyHostToDevice );
	hipBindTextureToArray( rT, cuArray );

	// part2. run kernel
	dim3	block( BLOCK_DIM, BLOCK_DIM ),
			grid( ceil( (float)w / BLOCK_DIM), ceil( (float)h / BLOCK_DIM) );
	Transpose_Texture<<< grid, block>>>( dImg, w, h );

	// part3. copy the data from device
	hipMemcpy( tImg, dImg, data_size, hipMemcpyDeviceToHost );

	// par4. release data
	hipUnbindTexture( rT );
	hipFreeArray( cuArray );
	hipFree( dImg );
}
